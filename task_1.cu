#include <hip/hip_runtime.h>

#include <cmath>
#include <iostream>
#include <fstream>
#include <vector>
#include <sys/time.h>

const float G = 6.67e-11;
const float dt = 0.01;
const float final_time = 100.0f;

__host__ void WriteState(FILE *out, int n, float time, float *positions) {
    fprintf(out, "%f", time);
    for (int index_point = 0; index_point < n; ++index_point) {
        fprintf(out, ",%f,%f", positions[2 * index_point],  positions[2 * index_point + 1]);
    }
    fprintf(out, "\n");
}

__global__ void CalcForces(int n, float *m, float *positions, float *totalF) {
    int first_point = blockDim.x * blockIdx.x + threadIdx.x;
    int second_point = blockDim.y * blockIdx.y + threadIdx.y;
    // Будем обновлять сразу обе противоположные силы
    if (first_point >= second_point || first_point >= n) {
        return;
    }
    float dist_x = positions[second_point * 2] - positions[first_point * 2];
    float dist_y = positions[second_point * 2 + 1] - positions[first_point * 2 + 1];
    float norm = powf(sqrtf(dist_x * dist_x + dist_y * dist_y), 3.0f) + 1e-12;
    float f_coef = G * m[first_point] * m[second_point] / norm;
    // Атомарно находим суммы сил, действующих на точки со стороны других точек. Получаем полные силы
    atomicAdd(&totalF[2 * first_point], dist_x * f_coef);
    atomicAdd(&totalF[2 * first_point + 1], dist_y * f_coef);
    atomicAdd(&totalF[2 * second_point], - dist_x * f_coef);
    atomicAdd(&totalF[2 * second_point + 1], - dist_y * f_coef);
}

__global__ void CalcState(int n, float* m, float *positions, float *V, float *F) {
    int point_index = blockDim.x * blockIdx.x + threadIdx.x;
    if (point_index >= n) {
        return;
    }
    int point_coord = threadIdx.y;
    int i = 2 * point_index + point_coord;
    // Обновляем позицию точки
    positions[i] += V[i] * dt;
    // Обновляем скорость
    V[i] += F[i] / m[point_index] * dt;
    // Зануляем силу для следующего вызова CalcForces
    F[i] = 0.0f;
}

int main() {
    int n;
    unsigned int thread_count;

    FILE *input = fopen("input_10_points.txt", "r");
    if (!input) {
        printf("Error opening input file");
        return -1;
    }

    fscanf(input, "%d", &n);

    printf("Count of thread: ");
    scanf("%d", &thread_count);

    unsigned int block_count = (n + thread_count - 1) / thread_count;

    float *m = (float*) malloc(n * sizeof(float));
    float *positions = (float*) malloc(2 * n * sizeof(float));
    float *V = (float*) malloc(2 * n * sizeof(float));

    for (int i = 0; i < n; ++i) {
        fscanf(input, "%f %f %f %f %f", &m[i], &positions[2 * i], &positions[2 * i + 1], &V[2 * i], &V[2 * i + 1]);
    }

    fclose(input);

    float* d_m, *d_positions, *d_V, *d_F;
    hipMalloc(&d_m, n * sizeof(float));
    hipMalloc(&d_positions, 2 * n * sizeof(float));
    hipMalloc(&d_V, 2 * n * sizeof(float));
    hipMalloc(&d_F, 2 * n * sizeof(float));

    hipMemcpy(d_m, m, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_positions, positions, 2 * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, 2 * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_F, 0.0f, 2 * n * sizeof(float));

    FILE *out = fopen("output.csv", "w");
    if (!out) {
        printf("Error with opening output file");
        return -1;
    }

    fprintf(out, "t");
    for (int index_point = 0; index_point < n; ++index_point) {
        fprintf(out, ",x_%d,y_%d", index_point,  index_point);
    }
    fprintf(out, "\n");

    WriteState(out, n, 0.0f, positions);

    int write_flag = 0;

    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);

    dim3 point_size = {thread_count, 2};
    dim3 forces_block_size = {thread_count, thread_count};

    for (float time = 0; time < final_time; time += dt) {
        CalcForces<<<block_count, forces_block_size>>>(n, d_m, d_positions, d_F);
        CalcState<<<block_count, point_size>>>(n, d_m, d_positions, d_V, d_F);
        hipMemcpy(positions, d_positions, 2 * n * sizeof(float), hipMemcpyDeviceToHost);
        if (write_flag) { WriteState(out, n, time + dt, positions); }
    }

    gettimeofday(&end_time, NULL);

    double time_taken = (end_time.tv_sec - start_time.tv_sec) * 1e6;
    time_taken = (time_taken + (end_time.tv_usec - start_time.tv_usec)) * 1e-6;

    printf("Time taken: %f seconds\n", time_taken);

    fclose(out);

    free(m);
    free(positions);
    free(V);

    hipFree(d_m);
    hipFree(d_positions);
    hipFree(d_V);
    hipFree(d_F);

    return 0;
}
